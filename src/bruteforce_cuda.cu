#include "hip/hip_runtime.h"
#include "bruteforce_cuda.hpp"
#include <algorithm>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <limits>
#include <linux/limits.h>

__device__ void swap(uint8_t *a, uint8_t *b) {
  uint8_t temp = *a;
  *a = *b;
  *b = temp;
}

__device__ bool next_permutation(uint8_t *first, uint8_t *last) {
  if (first == last)
    return false;
  uint8_t *i = last - 1;
  while (i > first && *(i - 1) >= *i)
    --i;
  if (i == first)
    return false;
  uint8_t *j = last - 1;
  while (*j <= *(i - 1))
    --j;
  swap(i - 1, j);
  uint8_t *k = last - 1;
  while (i < k)
    swap(i++, k--);
  return true;
}

__global__ void solve_kernel(uint16_t *distances, uint8_t *permutations,
                             uint16_t *results, std::size_t vertex_count,
                             uint16_t entry) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= (vertex_count - 1) * (vertex_count - 1))
    return;

  uint8_t second_vertex = tid % (vertex_count - 1);
  uint8_t third_vertex = (tid - second_vertex) / (vertex_count - 1);
  if (second_vertex == third_vertex) {
    results[tid] = std::numeric_limits<uint16_t>::max();
    return;
  }

  uint8_t *vertices = permutations + tid * sizeof(uint8_t) * vertex_count;
  vertices[0] = 0;
  vertices[1] = second_vertex + 1;
  vertices[2] = third_vertex + 1;

  int vertex_idx = 3;
  for (auto i = 1; i < vertex_count; i++)
    if (i != second_vertex + 1 && i != third_vertex + 1) {
      vertices[vertex_idx] = i;
      vertex_idx++;
    }

  uint16_t result = entry;
  do {
    uint16_t instance = 0;
    int i = 0;
    for (; i < vertex_count && instance < result; i++)
      instance += distances[vertices[i] * vertex_count + vertices[i + 1]];
    if (i == vertex_count && instance < result)
      result = instance;
  } while (next_permutation(vertices + 3, vertices + vertex_count));

  results[tid] = result;
}

uint16_t BruteforceCUDA::solve(Graph *graph,
                               std::optional<uint16_t> entry_value) {
  uint16_t entry = (entry_value.has_value())
                       ? entry_value.value()
                       : std::numeric_limits<uint16_t>::max();

  auto *host_distances = new uint16_t[graph->size() * graph->size()];
  for (auto from = 0; from < graph->size(); from++)
    for (auto to = 0; to < graph->size(); to++)
      host_distances[from * graph->size() + to] = graph->distance(from, to);

  uint16_t *device_distances;
  hipMalloc(&device_distances,
             sizeof(uint16_t) * graph->size() * graph->size());
  hipMemcpy(device_distances, host_distances,
             sizeof(uint16_t) * graph->size() * graph->size(),
             hipMemcpyHostToDevice);

  const std::size_t MAX_THREADS = (graph->size() - 1) * (graph->size() - 1);
  uint8_t *device_permutations;
  hipMalloc(&device_permutations,
             sizeof(uint8_t) * graph->size() * MAX_THREADS);

  uint16_t *device_results;
  hipMalloc(&device_results, sizeof(uint16_t) * MAX_THREADS);

  auto block_count = (MAX_THREADS + BLOCK_SIZE - 1) / BLOCK_SIZE;
  solve_kernel<<<block_count, BLOCK_SIZE>>>(device_distances,
                                            device_permutations, device_results,
                                            graph->size(), entry);
  hipError_t err = hipDeviceSynchronize();

  // Check for any errors during kernel execution
  if (err != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(err));
    exit(1);
  }

  auto *host_results = new uint16_t[MAX_THREADS];
  hipMemcpy(host_results, device_results, sizeof(uint16_t) * MAX_THREADS,
             hipMemcpyDeviceToHost);

  uint16_t result = *std::min_element(host_results, host_results + MAX_THREADS);
  delete[] host_distances;
  delete[] host_results;
  hipFree(device_distances);
  hipFree(device_permutations);
  hipFree(device_results);
  return result;
}
